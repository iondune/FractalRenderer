#include "hip/hip_runtime.h"

#include "CudaFractalRender.cuh"
#include "hip/hip_runtime.h"

#include <ionCore/ionUtils.h>




__global__ void InitKernel(SPixelState * States,  SFractalParams Params)
{
	cvec2u PixelCoordinates(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (PixelCoordinates.X >= Params.ScreenSize.X || PixelCoordinates.Y >= Params.ScreenSize.Y)
		return;

	SPixelState & State = States[PixelCoordinates.Y * Params.ScreenSize.X + PixelCoordinates.X];
	State.Counter = 0;
	State.Point = cvec2d();
	State.Iteration = 0;
	State.LastMax = 0;
	State.LastTotal = 0;
	State.FinalSum = 0;
	State.Finished = false;
	State.Calculated = false;
}

__global__ void HistogramKernel(SPixelState * States, u32 * Histogram, SFractalParams Params)
{
	cvec2u PixelCoordinates(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (PixelCoordinates.X >= Params.ScreenSize.X || PixelCoordinates.Y >= Params.ScreenSize.Y)
		return;

	SPixelState & State = States[PixelCoordinates.Y * Params.ScreenSize.X + PixelCoordinates.X];
	if (State.Finished)
		return;

	cvec2d Point = State.Point;
	u32 IterationCounter = State.Iteration;
	cvec2d StartPosition(PixelCoordinates.X / (f64) Params.ScreenSize.X, PixelCoordinates.Y / (f64) Params.ScreenSize.Y);
	StartPosition -= 0.5;
	StartPosition *= Params.Scale;
	StartPosition += Params.Center;

	while (Dot(Point, Point) < 256.0 && IterationCounter < Params.IterationMax)
	{
		Point = cvec2d(Point.X*Point.X - Point.Y*Point.Y + StartPosition.X, 2 * Point.X * Point.Y + StartPosition.Y);
		++ IterationCounter;
	}
	State.Iteration = IterationCounter;
	State.Point = Point;

	f64 ContinuousIterator = 0;
	if (IterationCounter < Params.IterationMax)
	{
		f64 Zn = sqrt(Dot(Point, Point));
		f64 Nu = log(log(Zn) / log(2.0)) / log(2.0);
		ContinuousIterator = IterationCounter + 1 - Nu;
		atomicAdd(Histogram + IterationCounter, 1);
		State.Finished = true;
	}
	else
	{
		ContinuousIterator = Params.IterationMax;
	}

	State.Counter = ContinuousIterator;
}

__device__ static void ColorFromHSV(f64 const hue, f64 const saturation, f64 value, u8 & r, u8 & g, u8 & b)
{
    int const hi = int(floor(hue / 60)) % 6;
    double const f = hue / 60 - floor(hue / 60);

    value = value * 255;
    int v = int(value);
    int p = int(value * (1 - saturation));
    int q = int(value * (1 - f * saturation));
    int t = int(value * (1 - (1 - f) * saturation));
	
    if (hi == 0)
	{
		r = v;
		g = t;
		b = p;
	}
    else if (hi == 1)
	{
		r = q;
		g = v;
		b = p;
	}
    else if (hi == 2)
	{
		r = p;
		g = v;
		b = t;
	}
    else if (hi == 3)
	{
		r = p;
		g = q;
		b = v;
	}
    else if (hi == 4)
	{
		r = t;
		g = p;
		b = v;
	}
    else
	{
		r = v;
		g = p;
		b = q;
	}
}

__global__ void DrawKernel(void * Image, SPixelState * States, u32 * Histogram, SFractalParams Params)
{
	cvec2u PixelCoordinates(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (PixelCoordinates.X >= Params.ScreenSize.X || PixelCoordinates.Y >= Params.ScreenSize.Y)
		return;
	
	SPixelState & State = States[PixelCoordinates.Y * Params.ScreenSize.X + PixelCoordinates.X];
	u32 const LastMax = State.LastMax;
	u32 const LastTotal = State.LastTotal;

	// Update Total
	u32 Total = LastTotal;
	for (u32 i = LastMax; i < Params.IterationMax; ++ i)
		Total += Histogram[i];
	State.LastMax = Params.IterationMax;
	State.LastTotal = Total;

	if (State.Finished)
	{
		f64 const Counter = State.Counter;
		u32 const Iteration = floor(Counter);
		f64 const Delta = Counter - (f64) Iteration;

		u32 Sum = 0;
		if (State.Calculated)
		{
			Sum = State.FinalSum;
		}
		else
		{
			for (u32 i = 0; i < Iteration; ++ i)
				Sum += Histogram[i];
			State.FinalSum = Sum;
			State.Calculated = true;
		}

		f64 Average = Sum / (f64) Total;
		f64 AverageOneUp = Average + Histogram[Iteration] / Total;
		Average = Average * (1 - Delta) + AverageOneUp * Delta;

		f64 const Hue = pow(Average, 8);
		//u8 r, g, b;
		//ColorFromHSV(Hue * 255, 1, 1, r, g, b);
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 0] = 0;
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 1] = (u8) (Hue * 255);
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 2] = (u8) ((1 - Hue) * 255);
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 3] = 255;
	}
	else
	{
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 0] = 0;
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 1] = 0;
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 2] = 0;
		((u8 *) Image)[PixelCoordinates.Y *  Params.ScreenSize.X * 4 + PixelCoordinates.X * 4 + 3] = 255;
		return;
	}
}


CudaFractalRenderer::CudaFractalRenderer(SFractalParams const & Params)
{
	u32 const ScreenSize = Params.ScreenSize.X * Params.ScreenSize.Y;
	u32 const StateSize = ScreenSize * sizeof(SPixelState);

	hipMalloc((void**) & DeviceStates, StateSize);

	DeviceHistogram = 0;
	Reset(Params);
}

CudaFractalRenderer::~CudaFractalRenderer()
{
	hipFree(DeviceStates);
	hipFree(DeviceHistogram);
}

void CudaFractalRenderer::Reset(SFractalParams const & Params)
{
	u32 const HistogramSize = (Params.IterationMax + 1) * sizeof(u32);
	u32 const BlockSize = 16;

	if (DeviceHistogram)
		hipFree(DeviceHistogram);
	hipMalloc((void**) & DeviceHistogram, HistogramSize);
	hipMemset(DeviceHistogram, 0, HistogramSize);

	dim3 const Grid(
		Params.ScreenSize.X / BlockSize + (Params.ScreenSize.X % BlockSize ? 1 : 0), 
		Params.ScreenSize.Y / BlockSize + (Params.ScreenSize.Y % BlockSize ? 1 : 0));
	dim3 const Block(BlockSize, BlockSize);
	InitKernel<<<Grid, Block>>>(DeviceStates, Params);

	IterationMax = 0;
}

void CudaFractalRenderer::Render(void * deviceBuffer, SFractalParams Params)
{
	u32 const BlockSize = 16;
	u32 IterationIncrement = 1;
	/*if (IterationMax < 2000)
		IterationIncrement = 2;
	if (IterationMax < 1000)
		IterationIncrement = 5;
	if (IterationMax < 500)
		IterationIncrement = 10;*/
	
	if (IterationMax < Params.IterationMax)
	{
		IterationMax = Min(IterationMax + IterationIncrement, Params.IterationMax);

		dim3 const Grid(
			Params.ScreenSize.X / BlockSize + (Params.ScreenSize.X % BlockSize ? 1 : 0), 
			Params.ScreenSize.Y / BlockSize + (Params.ScreenSize.Y % BlockSize ? 1 : 0));
		dim3 const Block(BlockSize, BlockSize);

		if (IterationMax <= Params.IterationMax)
		{
			Params.IterationMax = IterationMax;
			HistogramKernel<<<Grid, Block>>>(DeviceStates, DeviceHistogram, Params);
			DrawKernel<<<Grid, Block>>>(deviceBuffer, DeviceStates, DeviceHistogram, Params);
		}
	}
}
