#include "hip/hip_runtime.h"

#include "CudaFractalRender.cuh"
#include "hip/hip_runtime.h"

#include <ionCore/ionUtils.h>


__global__ void HistogramKernel(f64 * Counter, u32 * Histogram, SFractalParams Params)
{
	cvec2u PixelCoordinates(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	if (PixelCoordinates.X >= Params.ScreenSize.X || PixelCoordinates.Y >= Params.ScreenSize.Y)
		return;

	cvec2d StartPosition(PixelCoordinates.X / (f64) Params.ScreenSize.X, PixelCoordinates.Y / (f64) Params.ScreenSize.Y);
	StartPosition -= 0.5;
	StartPosition *= Params.Scale;
	StartPosition += Params.Center;

	cvec2d Point(0, 0);
	u32 IterationCounter = 0;
	while (Dot(Point, Point) < 256.0 && IterationCounter < Params.IterationMax)
	{
		Point = cvec2d(Point.X*Point.X - Point.Y*Point.Y + StartPosition.X, 2 * Point.X * Point.Y + StartPosition.Y);
		++ IterationCounter;
	}

	f64 ContinuousIterator = 0;
	if (IterationCounter < Params.IterationMax)
	{
		f64 Zn = sqrt(Dot(Point, Point));
		f64 Nu = log(log(Zn) / log(2.0)) / log(2.0);
		ContinuousIterator = IterationCounter + 1 - Nu;
	}
	else
		ContinuousIterator = Params.IterationMax;
	atomicAdd(Histogram + IterationCounter, 1);
	Counter[PixelCoordinates.Y * Params.ScreenSize.X + PixelCoordinates.X] = ContinuousIterator;
}

__device__ static void ColorFromHSV(f64 const hue, f64 const saturation, f64 value, u8 & r, u8 & g, u8 & b)
{
    int const hi = int(floor(hue / 60)) % 6;
    double const f = hue / 60 - floor(hue / 60);

    value = value * 255;
    int v = int(value);
    int p = int(value * (1 - saturation));
    int q = int(value * (1 - f * saturation));
    int t = int(value * (1 - (1 - f) * saturation));
	
    if (hi == 0)
	{
		r = v;
		g = t;
		b = p;
	}
    else if (hi == 1)
	{
		r = q;
		g = v;
		b = p;
	}
    else if (hi == 2)
	{
		r = p;
		g = v;
		b = t;
	}
    else if (hi == 3)
	{
		r = p;
		g = q;
		b = v;
	}
    else if (hi == 4)
	{
		r = t;
		g = p;
		b = v;
	}
    else
	{
		r = v;
		g = p;
		b = q;
	}
}

__global__ void DrawKernel(u8 * Image, f64 * Counter, u32 * Histogram, SFractalParams Params)
{
	cvec2u PixelCoordinates(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	
	if (PixelCoordinates.X >= Params.ScreenSize.X || PixelCoordinates.Y >= Params.ScreenSize.Y)
		return;

	if (Counter[PixelCoordinates.Y * Params.ScreenSize.X + PixelCoordinates.X] == Params.IterationMax)
	{
		Image[PixelCoordinates.Y *  Params.ScreenSize.X * 3 + PixelCoordinates.X * 3 + 0] = 0;
		Image[PixelCoordinates.Y *  Params.ScreenSize.X * 3 + PixelCoordinates.X * 3 + 1] = 0;
		Image[PixelCoordinates.Y *  Params.ScreenSize.X * 3 + PixelCoordinates.X * 3 + 2] = 0;
		return;
	}

	u32 iteration = floor(Counter[PixelCoordinates.Y * Params.ScreenSize.X + PixelCoordinates.X]);
	f64 total = 0;
	for (u32 i = 0; i < Params.IterationMax; ++ i)
		total += Histogram[i];

	f64 hue = 0;
	for (u32 i = 0; i < iteration; ++ i)
		hue += Histogram[i] / total;
	f64 oneuphue = hue + Histogram[iteration] / total;

	f64 delta = Counter[PixelCoordinates.Y * Params.ScreenSize.X + PixelCoordinates.X] - (f64) iteration;
	hue = hue * (1 - delta) + oneuphue * delta;

	u8 r, g, b;
	f64 hueit = pow(hue, 8);
	ColorFromHSV(hueit * 255, 1, 1, r, g, b);
	Image[PixelCoordinates.Y *  Params.ScreenSize.X * 3 + PixelCoordinates.X * 3 + 0] = 0;
	Image[PixelCoordinates.Y *  Params.ScreenSize.X * 3 + PixelCoordinates.X * 3 + 1] = (u8) (hueit * 255);
	Image[PixelCoordinates.Y *  Params.ScreenSize.X * 3 + PixelCoordinates.X * 3 + 2] = (u8) ((1 - hueit) * 255);
}

u8 const * CudaRenderFractal(SFractalParams const & Params)
{
	u32 const ScreenSize = Params.ScreenSize.X * Params.ScreenSize.Y;
	u32 const ImageSize = ScreenSize * 3;
	u32 const CounterSize = ScreenSize * sizeof(f64);
	u32 const HistogramSize = (Params.IterationMax + 1) * sizeof(u32);

	u8 * HostImage = new u8[ImageSize];

	u8 * DeviceImage; hipMalloc((void**) & DeviceImage, ImageSize);
	f64 * DeviceCounter; hipMalloc((void**) & DeviceCounter, CounterSize);
		hipMemset(DeviceCounter, 0, CounterSize);
	u32 * DeviceHistogram; hipMalloc((void**) & DeviceHistogram, HistogramSize);
		hipMemset(DeviceHistogram, 0, HistogramSize);

	u32 const BlockSize = 16;
	dim3 const Grid(
		Params.ScreenSize.X / BlockSize + (Params.ScreenSize.X % BlockSize ? 1 : 0), 
		Params.ScreenSize.Y / BlockSize + (Params.ScreenSize.Y % BlockSize ? 1 : 0));
	dim3 const Block(BlockSize, BlockSize);
	HistogramKernel<<<Grid, Block>>>(DeviceCounter, DeviceHistogram, Params);
	DrawKernel<<<Grid, Block>>>(DeviceImage, DeviceCounter, DeviceHistogram, Params);

	hipMemcpy(HostImage, DeviceImage, ImageSize * sizeof(u8), hipMemcpyDeviceToHost);
	hipFree(DeviceImage);
	hipFree(DeviceCounter);
	hipFree(DeviceHistogram);

	return HostImage;
}
